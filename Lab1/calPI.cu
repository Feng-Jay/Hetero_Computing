
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>

#define PerThread 2048*2//每个线程计算多少个i
#define N 64*256*PerThread//积分计算PI总共划分为这么多项相加
#define BlockNum 64 //block的数量
#define ThreadNum 256 //每个block中threads的数量

__global__ void Gpu_calPI(double* Gpu_list)
{
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;
    int begin=tid*PerThread;
    int end=begin+PerThread-1;
    double temp=0;
    for(int i=begin;i<end;i++){
        temp+=4.0/(1+((i+0.5)/(N))*((i+0.5)/(N)));
    }
    Gpu_list[tid]=temp;
}

int main(void)
{
    double * cpu_list;
    double * Gpu_list;
    double outcome=0;
    cpu_list=(double*)malloc(sizeof(double)*BlockNum*ThreadNum);
    hipMalloc((void**)&Gpu_list,sizeof(double)*BlockNum*ThreadNum);
    // dim3 blocksize=dim3(1,ThreadNum);
    // dim3 gridsize=dim3(1,BlockNum);
    double begin=clock();
    Gpu_calPI<<<BlockNum,ThreadNum>>>(Gpu_list);

    hipMemcpy(cpu_list,Gpu_list,sizeof(double)*BlockNum*ThreadNum,hipMemcpyDeviceToHost);
    for(int i=0;i<BlockNum*ThreadNum;i++){
        outcome+=cpu_list[i];
    }
    outcome=outcome/(N);
    double end=clock();
    printf("Cu1: N=%d, outcome=%.10f, time =%.10f\n",N,outcome,(end-begin)/(CLOCKS_PER_SEC));
    // printf("block x=%d,y=%d\n",blocksize.x,blocksize.y);
    // printf("grid x=%d,y=%d\n",gridsize.x,gridsize.y);
    
}