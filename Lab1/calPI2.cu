
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define PerThread 1024*16//每个线程计算多少个i
#define N 64*256*1024*16//积分计算PI总共划分为这么多项相加
#define BlockNum 64 //block的数量
#define ThreadNum 256 //每个block中threads的数量

__global__ void Gpu_calPI(double* Gpu_list)
{
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;
    int begin=tid*PerThread+1;
    int end=begin+PerThread;
    double temp=0;
    int flag=1;
    for(int i=begin;i<end;i++){
        temp+=flag*(1.0/(2*i-1));
        flag=flag*(-1);
    }
    Gpu_list[tid]=temp;
}

int main(void)
{
    double * cpu_list;
    double * Gpu_list;
    double outcome=0;
    cpu_list=(double*)malloc(sizeof(double)*BlockNum*ThreadNum);
    hipMalloc((void**)&Gpu_list,sizeof(double)*BlockNum*ThreadNum);
    // dim3 blocksize=dim3(1,ThreadNum);
    // dim3 gridsize=dim3(1,BlockNum);
    double begin=clock();
    Gpu_calPI<<<BlockNum,ThreadNum>>>(Gpu_list);

    hipMemcpy(cpu_list,Gpu_list,sizeof(double)*BlockNum*ThreadNum,hipMemcpyDeviceToHost);
    for(int i=0;i<BlockNum*ThreadNum;i++){
        outcome+=cpu_list[i];
    }
    outcome=4*outcome;
    double end=clock();
    printf("Cu2: N=%d, outcome=%.10f,time spend %.10f\n",N,outcome,(end-begin)/(CLOCKS_PER_SEC));
    // printf("block x=%d,y=%d\n",blocksize.x,blocksize.y);
    // printf("grid x=%d,y=%d\n",gridsize.x,gridsize.y);
    
}