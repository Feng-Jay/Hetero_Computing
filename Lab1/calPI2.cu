#include <stdio.h>
#include <hip/hip_runtime.h>


#define PerThread 1024
#define N 64*256*PerThread//积分计算PI总共划分为这么多项相加
#define BlockNum 64 //block的数量
#define ThreadNum 256 //每个block中threads的数量

__global__ void Gpu_calPI(double* Gpu_list)
{
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;
    int begin=tid*PerThread+1;
    int end=begin+PerThread;
    double temp=0;
    int flag=1;
    for(int i=begin;i<end;i++){
        temp+=flag*(1.0/(2*i-1));
        flag=flag*(-1);
    }
    Gpu_list[tid]=temp;
}

int main(void)
{
    double * cpu_list;
    double * Gpu_list;
    double outcome=0;
    cpu_list=(double*)malloc(sizeof(double)*BlockNum*ThreadNum);
    hipMalloc((void**)&Gpu_list,sizeof(double)*BlockNum*ThreadNum);
    // dim3 blocksize=dim3(1,ThreadNum);
    // dim3 gridsize=dim3(1,BlockNum);

    Gpu_calPI<<<BlockNum,ThreadNum>>>(Gpu_list);

    hipMemcpy(cpu_list,Gpu_list,sizeof(double)*BlockNum*ThreadNum,hipMemcpyDeviceToHost);
    for(int i=0;i<BlockNum*ThreadNum;i++){
        outcome+=cpu_list[i];
    }
    outcome=4*outcome;
    printf("outcome=%.10f\n",outcome);
    // printf("block x=%d,y=%d\n",blocksize.x,blocksize.y);
    // printf("grid x=%d,y=%d\n",gridsize.x,gridsize.y);
    
}