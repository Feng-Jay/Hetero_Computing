
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define PerThread 1024*4*8//每个线程计算多少个i
#define N 64*256*1024*4//积分计算PI总共划分为这么多项相加
#define BlockNum 32 //block的数量
#define ThreadNum 64 //每个block中threads的数量

__global__ void Gpu_calPI(double* Gpu_list)
{
    __shared__ double cache[ThreadNum];//每个block共享一个shared memory.
    int cacheIdx=threadIdx.x;
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;
    int begin=tid*PerThread;
    int end=begin+PerThread-1;
    double temp=0;
    for(int i=begin;i<end;i++){
        temp+=4.0/(1+((i+0.5)/(N))*((i+0.5)/(N)));
    }
    cache[cacheIdx]=temp;
    __syncthreads();//同步

    int i=blockDim.x/2;
    while(i!=0){
        if(cacheIdx<i) cache[cacheIdx]+=cache[cacheIdx+i];
        __syncthreads();//同步
        i=i/2;
    }

    if(cacheIdx==0){
        Gpu_list[blockIdx.x]=cache[0];
    }
}

int main(void)
{
    double * cpu_list;
    double * Gpu_list;
    double outcome=0;
    cpu_list=(double*)malloc(sizeof(double)*BlockNum);
    hipMalloc((void**)&Gpu_list,sizeof(double)*BlockNum);
    // dim3 blocksize=dim3(1,ThreadNum);
    // dim3 gridsize=dim3(1,BlockNum);
    double begin = clock();
    Gpu_calPI<<<BlockNum,ThreadNum>>>(Gpu_list);

    hipMemcpy(cpu_list,Gpu_list,sizeof(double)*BlockNum,hipMemcpyDeviceToHost);
    for(int i=0;i<BlockNum;i++){
        outcome+=cpu_list[i];
    }
    outcome=outcome/(N);
    double end=clock();
    printf("Scu1: N=%d, outcome=%.10f,time spend %.10f\n",N,outcome,(end-begin)/(CLOCKS_PER_SEC));
    // printf("block x=%d,y=%d\n",blocksize.x,blocksize.y);
    // printf("grid x=%d,y=%d\n",gridsize.x,gridsize.y);
    
}