
#include <hip/hip_runtime.h>
#include <stdio.h>

#define PerThread 1024//每个线程计算多少个i
#define N 64*256*PerThread//积分计算PI总共划分为这么多项相加
#define BlockNum 64 //block的数量
#define ThreadNum 256 //每个block中threads的数量

__global__ void Gpu_calPI(double* Gpu_list)
{
    __shared__ double cache[ThreadNum];//每个block共享一个shared memory.
    int cacheIdx=threadIdx.x;
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;
    int begin=tid*PerThread;
    int end=begin+PerThread-1;
    double temp=0;
    for(int i=begin;i<end;i++){
        temp+=4.0/(1+((i+0.5)/(N))*((i+0.5)/(N)));
    }
    cache[cacheIdx]=temp;
    __syncthreads();//同步

    int i=blockDim.x/2;
    while(i!=0){
        if(cacheIdx<i) cache[cacheIdx]+=cache[cacheIdx+i];
        __syncthreads();//同步
        i=i/2;
    }

    if(cacheIdx==0){
        Gpu_list[blockIdx.x]=cache[0];
    }
}

int main(void)
{
    double * cpu_list;
    double * Gpu_list;
    double outcome=0;
    cpu_list=(double*)malloc(sizeof(double)*BlockNum);
    hipMalloc((void**)&Gpu_list,sizeof(double)*BlockNum);
    // dim3 blocksize=dim3(1,ThreadNum);
    // dim3 gridsize=dim3(1,BlockNum);

    Gpu_calPI<<<BlockNum,ThreadNum>>>(Gpu_list);

    hipMemcpy(cpu_list,Gpu_list,sizeof(double)*BlockNum,hipMemcpyDeviceToHost);
    for(int i=0;i<BlockNum;i++){
        outcome+=cpu_list[i];
    }
    outcome=outcome/(N);
    printf("outcome=%.10f\n",outcome);
    // printf("block x=%d,y=%d\n",blocksize.x,blocksize.y);
    // printf("grid x=%d,y=%d\n",gridsize.x,gridsize.y);
    
}