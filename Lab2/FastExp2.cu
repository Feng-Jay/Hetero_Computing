
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define M 32//矩阵维度
#define Exp 4096//计算的幂次

__constant__ const int gpu_m=M;
__constant__ const int gpu_exp=Exp;

__global__ void Gpu_FastExp(float *gpu_martix,float* gpu_res)
{
    // printf("%d ", threadIdx.y)
    __shared__ float temp[gpu_m*gpu_m];
    float res[gpu_m];
    int tid=threadIdx.y;
    temp[tid]=gpu_martix[tid];
    __syncthreads();
    for(int i=1;i<gpu_exp;i*=2){
        for(int k=0;k<M;k++)
        {
            res[k]=0;
            for(int j=0;j<M;j++){
                res[k]+=temp[tid*M+j]*temp[j*M+tid];
            }
        }
        for(int i=0;i<M;i++){
        gpu_res[tid*M+i]=res[i];
        temp[tid*M+i]=res[i];
        }
        __syncthreads();
    }
}

int main()
{
    float * cpu_martix;
    float * cpu_res;
    float* gpu_martix;
    float * gpu_res;
    hipHostMalloc((void**)&cpu_martix,M*M*sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&cpu_res,M*M*sizeof(float), hipHostMallocDefault);
    hipMalloc((void**)&gpu_res,M*M*sizeof(float));
    hipMalloc((void**)&gpu_martix,M*M*sizeof(float));
    for(int i=0;i<M;i++){
        for(int j=0;j<M;j++){
            cpu_martix[i*M+j] = 1.0/M;
        }
    }
    printf("init elem=%f",1.0/M);
    hipMemcpy(gpu_martix,cpu_martix,M*M*sizeof(float),hipMemcpyHostToDevice);
    float begin=clock();
    dim3 threads(1,M);
    // printf("一切正常\n");
    Gpu_FastExp<<<1,threads>>>(gpu_martix,gpu_res);
    hipDeviceSynchronize();
    // printf("一切正常\n");
    hipMemcpy(cpu_res,gpu_res,M*M*sizeof(float),hipMemcpyDeviceToHost);
    float end=clock();
    printf("result = %.10f time spend=%.10f\n",cpu_res[0],(end-begin)/CLOCKS_PER_SEC);
}
