
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define M 32//矩阵维度
#define Exp 64//计算的幂次

__constant__ const int gpu_m=M;
__constant__ const int gpu_exp=Exp;

__global__ void Gpu_FastExp(float *gpu_martix,float* gpu_res)
{
    // printf("%d ", threadIdx.y)
    __shared__ float temp[gpu_m*gpu_m];
    int tid=threadIdx.y;
    temp[tid]=gpu_martix[tid];
    __syncthreads();
    for(int i=1;i<gpu_exp;i*=2){
        float res=0.0;
        for(int j=0;j<M;j++){
            res+=temp[(tid/M)*M+j]*temp[j*M+(tid%M)];
        }
        gpu_res[tid]=res;
        temp[tid]=res;
        __syncthreads();
    }


}

int main()
{
    float * cpu_martix;
    float * cpu_res;
    float* gpu_martix;
    float * gpu_res;
    hipHostMalloc((void**)&cpu_martix,M*M*sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&cpu_res,M*M*sizeof(float), hipHostMallocDefault);
    hipMalloc((void**)&gpu_res,M*M*sizeof(float));
    hipMalloc((void**)&gpu_martix,M*M*sizeof(float));
    for(int i=0;i<M;i++){
        for(int j=0;j<M;j++){
            cpu_martix[i*M+j] = 1.0/M;
        }
    }
    printf("init elem=%f",1.0/M);
    hipMemcpy(gpu_martix,cpu_martix,M*M*sizeof(float),hipMemcpyHostToDevice);
    float begin=clock();
    dim3 threads(1,M*M);
    // printf("一切正常\n");
    Gpu_FastExp<<<1,threads>>>(gpu_martix,gpu_res);
    hipDeviceSynchronize();
    printf("一切正常\n");
    hipMemcpy(cpu_res,gpu_res,M*M*sizeof(float),hipMemcpyDeviceToHost);
    float end=clock();
    printf("result = %.10f time spend=%f\n",cpu_res[0],(end-begin)/CLOCKS_PER_SEC);
}
