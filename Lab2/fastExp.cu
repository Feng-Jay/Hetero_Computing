
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 99//矩阵的N次方
// 99=64+32+2+1
#define BlockNum 1//block的数量
#define ThreadNum 4 //每个block中threads的数量
#define m 32//每个行有多少个元素，即矩阵的维度

__global__ void Gpu_func(double* gpu_martix, double* gpu_res,int* exp)
{
   int tid=threadIdx.x;
   int index=tid*m*m;
   printf("tid=%d exp=%d\n",tid,exp[tid]);
   __shared__ double cache[m*m*3];
   for(int i=0;i<exp[tid];i++){
      for(int p=0;p<m;p++){
         for(int q=0;q<m;q++){
            for(int k=0;k<m;k++){
               gpu_res[p*m+q]+=gpu_martix[p*m+k]*gpu_martix[k*m+q];
            }
         }
      }
      for(int i=0;i<m;i++){
         for(int j=0;j<m;j++){
            gpu_martix[i*m+j]=gpu_res[i*m+j];
            gpu_res[i*m+j]=0;
         }
      }
   }
   if(tid!=0){
    for(int i=0;i<m;i++){
         for(int j=0;j<m;j++){
            cache[(tid-1)*m*m+i*m+j]=gpu_martix[i*m+j];
            __syncthreads();
         }
      }
   }
   if(tid==0){
      for(int num=0;num<3;num++){
         for(int i=0;i<m;i++){
            for(int j=0;j<m;j++){
               for(int k=0;k<m;k++){
                  gpu_res[i*m+j]+=gpu_martix[i*m+k]*cache[num*m*m+k*m+j]; 
               }
            }
         }
         if(num<2){
            for(int i=0;i<m;i++){
               for(int j=0;j<m;j++){
                  gpu_martix[i*m+j]=gpu_res[i*m+j];
                  gpu_res[i*m+j]=0;
               }
            }
         }
      }
   }

}

int 
main()
{
   double* cpu_martix;
   double* gpu_martix;
   double* cpu_res;
   double* gpu_res;
   int* gpu_exp;
   cpu_martix=(double*)malloc(sizeof(double)*m*m);
   cpu_res=(double*)malloc(sizeof(double)*m*m);
   for(int i=0;i<m*m;i++){
       cpu_martix[i]=1.0/(m);
       cpu_res[i]=0;
   }
   int exp[4]={6,5,1,0};
   printf("init elem is %f",1.0/(m));
   printf("\n");
   double begin=clock();
   hipMalloc((void**)&gpu_martix,sizeof(double)*m*m);
   hipMalloc((void**)&gpu_res,sizeof(double)*m*m);
   hipMalloc((void**)&gpu_exp,sizeof(int)*4);
   hipMemcpy(gpu_martix,cpu_martix,sizeof(double)*m*m,hipMemcpyHostToDevice);
   hipMemcpy(gpu_res,cpu_res,sizeof(double)*m*m,hipMemcpyHostToDevice);
   hipMemcpy(gpu_exp,exp,sizeof(int)*4,hipMemcpyHostToDevice);

   Gpu_func<<<BlockNum,ThreadNum>>>(gpu_martix,gpu_res,gpu_exp);
   hipMemcpy(cpu_res,gpu_res,sizeof(double)*m*m,hipMemcpyDeviceToHost);
   double end=clock();
    printf("spend %.10f s\n",(end-begin)/(CLOCKS_PER_SEC));
    printf("here is outcome martix......................\n");
    printf("%.20f\n",cpu_res[0]);
}