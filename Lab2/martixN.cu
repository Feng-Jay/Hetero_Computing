
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 64//矩阵的N次方
#define BlockNum 8//block的数量
#define ThreadNum 64 //每个block中threads的数量
#define m 32//每个行有多少个元素，即矩阵的维度

__global__ void Gpu_martixN(double* Gpu_martix, double* Gpu_res)
{
    //每个GPU核函数计算矩阵的一行
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;
    double temp[m];
    for(int i=0;i<m;i++) temp[i]=0;

    for(int exp=0;exp<N;exp++){
        for(int i=0;i<m;i++){
            for(int j=0;j<m;j++){
                temp[i]+=Gpu_res[tid*m+j]*Gpu_martix[j*m+i];
            }//算完1个
        }//算完1行
        for(int i=0;i<m;i++) {Gpu_res[tid*m+i]=temp[i];temp[i]=0;}
    }
}

int main()
{
    double* Cpu_martix;
    double* Cpu_res;
    double* Gpu_martix;
    double* Gpu_res;
    Cpu_martix=(double*)malloc(sizeof(double)*m*m);
    Cpu_res=(double*)malloc(sizeof(double)*m*m);
    //初始化矩阵
    for(int i=0;i<m*m;i++){
       Cpu_martix[i]=1.0/(m);
       Cpu_res[i]=1.0/(m);
    }
    printf("init elem is %f",1.0/(m));
    printf("\n");
    hipMalloc((void**)&Gpu_martix,sizeof(double)*m*m);
    hipMalloc((void**)&Gpu_res,sizeof(double)*m*m);
    hipMemcpy(Gpu_martix,Cpu_martix,sizeof(double)*m*m,hipMemcpyHostToDevice);
    hipMemcpy(Gpu_res,Cpu_res,sizeof(double)*m*m,hipMemcpyHostToDevice);
    printf("begin exec\n");
    double begin=clock();
    Gpu_martixN<<<BlockNum,ThreadNum>>>(Gpu_martix,Gpu_res);
    double end=clock();
    hipMemcpy(Cpu_res,Gpu_res,sizeof(double)*m*m,hipMemcpyDeviceToHost);
    printf("spend %.10f s\n",(end-begin)/(CLOCKS_PER_SEC));
    printf("here is outcome martix......................\n");
    printf("%f\n",Cpu_res[0]);
}