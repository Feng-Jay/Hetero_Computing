#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 64*256*1024//积分计算PI总共划分为这么多项相加
#define BlockNum 64 //block的数量
#define ThreadNum 256 //每个block中threads的数量

__global__ void Gpu_calPI(double* Gpu_list)
{
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;
    int begin=tid*1024;
    int end=begin+1024-1;
    double temp=0;
    for(int i=begin;i<end;i++){
        temp+=4.0/(1+((i+0.5)/(N))*((i+0.5)/(N)));
    }
    Gpu_list[tid]=temp;
}

int main(void)
{
    double * cpu_list;
    double * Gpu_list;
    double outcome=0;
    cpu_list=(double*)malloc(sizeof(double)*BlockNum*ThreadNum);
    hipMalloc((void**)&Gpu_list,sizeof(double)*BlockNum*ThreadNum);
    // dim3 blocksize=dim3(1,ThreadNum);
    // dim3 gridsize=dim3(1,BlockNum);

    Gpu_calPI<<<BlockNum,ThreadNum>>>(Gpu_list);

    hipMemcpy(cpu_list,Gpu_list,sizeof(double)*BlockNum*ThreadNum,hipMemcpyDeviceToHost);
    for(int i=0;i<BlockNum*ThreadNum;i++){
        outcome+=cpu_list[i];
    }
    outcome=outcome/(N);
    printf("outcome=%.10f\n",outcome);
    // printf("block x=%d,y=%d\n",blocksize.x,blocksize.y);
    // printf("grid x=%d,y=%d\n",gridsize.x,gridsize.y);
    
}